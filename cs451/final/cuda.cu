#include "hip/hip_runtime.h"



// Kernel function where each thread does a single element
__global__ void kernel_fxn(float* A, float* B, int N) {
    // Get element to do
    int e = blockIdx.x * blockDim.x + threadIdx.x;
    if (e >= N * N)
        return;

    //
    B[e] = 4 * A[e];
}

// Kernel function where each thread does a single row
__global__ void kernel_fxn2(float* A, float* B, int N) {
    // Get row to do
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= N)
        return;

    for (int i = 0; i < N; i++)
        B[row * N + i] = 4 * A[row * N + i];
}


// Host function
int main(int argc, char** argv) {
    // A and B are the matricies
    float* A, B;
    // let N be the dimension of the matricies
    int N;
    get_params(A, B, &N);

    // Reserve space on GPU
    float* gpu_A, * gpu_B;
    hipMalloc((void**) &gpu_A, N * N * sizeof(float));
    hipMalloc((void**) &gpu_B, N * N * sizeof(float));

    // Send problem to device
    hipMemcpy((void*) gpu_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Make device do the problem
    kernel_fxn<<n_blocks, n_threads_per_block>>>(gpu_A, gpu_B, N);

    // Pull solution from the device
    hipMemcpy((void*) B, gpu_B, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Solution now stored in B
}