/* Matrix normalization.
 * Compile with "gcc matrixNorm.c"
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

// Matrix dimension
int N;

// Cuda variables
int n_blocks = 16;
int n_threads_per_block = 32;

/* Initialize A and B*/
void initialize_inputs(int argc, char** argv, float*& A, float*& B) {
    // User requested specific number of threads
    if (argc <= 1) {
        printf("USAGE: %s <N> <blocks> <threads per block> <seed>\n", argv[0]);
        exit(1);
    }

    // Set dimension
    if (argc > 1)
        N = atoi(argv[1]);

    // Set number of blocks
    if (argc > 2) {
        n_blocks = atoi(argv[2]);
    }

    // Set threads per block
    if (argc > 3) {
        n_threads_per_block = atoi(argv[3]);
    }

    // Set seed
    if (argc > 4)
        srand(atoi(argv[4]));
    else
        srand((unsigned)time(NULL));

    // Allocate space for the matricies
    A = (float*) malloc(N * N * sizeof(float));
    B = (float*) malloc(N * N * sizeof(float));

    // Initialize the matrix with random values
    for (int row = 0; row < N; row++)
        for (int col = 0; col < N; col++) {
            A[row * N + col] = (float)rand() / 32768.0;
            B[row * N + col] = 0.0;
        }
}

/// Print a matrix's content for debugging
void print_matrix(float* m) {
    printf("[");
    for (int row = 0; row < N; row++) {
        printf("\n\t");
        for (int col = 0; col < N; col++)
            printf("%10.5f\t", m[row * N + col]);
    }
    printf("]\n");
}

 /* Kernel function */
__global__ void matrixNorm(float* A, float* B, int N) {
    // Calculate column number
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Maybe some wasted threads
    if (col < N) {
        int row;         // Row index for loops
        float mu, sigma; // Mean and Standard Deviation

        // Calculate mean for column
        mu = 0.0;
        for (row = 0; row < N; ++row)
            mu += A[row * N + col];
            mu /= N;
        __syncthreads();

        // Calculate standard deviation for the column
        sigma = 0.0;
        for (row = 0; row < N; ++row)
            sigma += powf(A[col * N + row] - mu, 2.0);
        sigma /= N;
        sigma = sqrt(sigma);
        __syncthreads();

        // Normalize column
        for (row = 0; row < N; ++row)
            if (sigma == 0.0)
                B[row * N + col] = 0.0;
            else
                B[row * N + col] = (A[col * N + row] - mu) / sigma;
    }
}

int main(int argc, char** argv) {
    /* Timing variables */
    struct timeval start, stop;  /* Elapsed times using gettimeofday() */
    struct timeval norm_start, norm_stop; // time values for the normalization algorithm
    struct timezone tzdummy;
    unsigned long long runtime;

    /* Initialize A and B */
    float* A, * B;
    initialize_inputs(argc, argv, A, B);

    /* Start Clock */
    gettimeofday(&start, &tzdummy);
    // Create buffers on device
    float* gpu_A, * gpu_B;
    hipMalloc((void**) &gpu_A, N * N * sizeof(float));
    hipMalloc((void**) &gpu_B, N * N * sizeof(float));

    // Send problem to device
    hipMemcpy((void*) gpu_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*) gpu_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    /* Matrix Normalization */
    gettimeofday(&norm_start, &tzdummy);
    matrixNorm<<<n_blocks, n_threads_per_block>>>(gpu_A, gpu_B, N);
    gettimeofday(&norm_stop, &tzdummy);

    // Pull result from the device
    hipMemcpy((void*) B, gpu_B, N * N * sizeof(float), hipMemcpyDeviceToHost);

    /* Calculate runtimes */
    gettimeofday(&stop, &tzdummy);
    runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);
    unsigned long long norm_time = (unsigned long long)
	    (norm_stop.tv_sec - norm_start.tv_sec) * 1000000 + (norm_stop.tv_usec - norm_start.tv_usec);

    /* Display timing results */
    printf("Runtime = %g ms.\n", (float)runtime/(float)1000);
    printf("Normalization time = %g ms.\n", (float)norm_time/(float)1000);

    // Debug for small N
    if (N <= 20) {
        print_matrix(A);
        print_matrix(B);
    }

    // Cleanup and exit
    free(A);
    free(B);
    hipFree(gpu_A);
    hipFree(gpu_B);
    return 0;
}
