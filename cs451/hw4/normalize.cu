#include "hip/hip_runtime.h"
/* Matrix normalization.
 * Compile with "gcc matrixNorm.c"
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <time.h>
 #include <sys/time.h>
 #include <math.h>

 /* Program Parameters */
 #define N 6000  /* Matrix size */

 /* Matrices */
 volatile float A[N][N], B[N][N];


 /* Initialize A and B*/
void initialize_inputs() {
    int row, col;

    srand((unsigned)time(NULL));
    for (row = 0; row < N; row++)
        for (col = 0; col < N; col++) {
            A[row][col] = (float)rand() / 32768.0;
            B[row][col] = 0.0;
        }

}

 /* Kernel function */

__global__ void matrixNorm(float* A, float* B) {
     int col = blockIdx.x * blockDim.x + threadIdx.x;
     __shared__ int row;
     __shared__ float mu, sigma; // Mean and Standard Deviation

    // Clamp to number of cols
    if (col < N) {
        // Calculate mean for column
        mu = 0.0;
        for (row = 0; row < N; ++row)
            mu += A[row * N + col];
            mu /= N;
        __syncthreads();

        // Calculate standard deviation for the column
        sigma = 0.0;
        for (row = 0; row < N; ++row)
            sigma += powf(A[col * N + row] - mu, 2.0);
        sigma /= N;
        sigma = sqrt(sigma);
        __syncthreads();

        // Normalize column
        for (row = 0; row < N; ++row)
            if (sigma == 0.0)
                B[row * N + col] = 0.0;
            else
                B[row * N + col] = (A[col * N + row] - mu) / sigma;
    }
}

int main(int argc, char **argv) {
    /* Timing variables */
    struct timeval start, stop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    unsigned long long runtime;

    /* Initialize A and B */
    initialize_inputs();

    /* Start Clock *//*
    printf("\n---------------------------------------------\n");
    printf("Matrix size N = %d", N);
    printf("\nStarting clock.\n\n");*/
    gettimeofday(&start, &tzdummy);

    // Create buffers on device
    float* gpu_A, * gpu_b;
    hipMalloc((void**) &gpu_A, sizeof(float) * N * N);
    hipMalloc((void**) &gpu_B, sizeof(float) * N * N);

    // Send problem to device
    hipMemcpy(A, gpu_A, sizeof(float) * N * N, hipMemcpyHostToDevice);

    /* Matrix Normalization */
    matrixNorm<<<numBlocks,numThreadsPerBlock>>>(gpu_A, gpu_B);

    // Pull result from the device
    hipMemcpy((void*)B, gpu_B, sizeof(float) * N * N, hipMemcpyDeviceToHost);

    /* Stop Clock */
    gettimeofday(&stop, &tzdummy);
    runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);

    /* Display timing results */
/*    printf("Runtime = %g ms.\n", (float)runtime/(float)1000);
    printf("\nStopped clock.");
    printf("\n---------------------------------------------\n");*/

    return 0;
}
