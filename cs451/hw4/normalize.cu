/* Matrix normalization.
 * Compile with "gcc matrixNorm.c"
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

// Matrix dimension
int N;

// Cuda variables
int n_blocks = 16;
int n_threads_per_block = 32;

/* Initialize A and B*/
void initialize_inputs(int argc, char** argv, float*& A, float*& B) {
    // User requested specific number of threads
    if (argc <= 1) {
        printf("USAGE: %s <N> <blocks> <threads per block> <seed>\n", argv[0]);
        exit(1);
    }

    // Set dimension
    if (argc > 1)
        N = atoi(argv[1]);

    // Set number of blocks
    if (argc > 2) {
        n_blocks = atoi(argv[2]);
    }

    // Set threads per block
    if (argc > 3) {
        n_threads_per_block = atoi(argv[2]);
    }

    // Set seed
    if (argc > 4)
        srand(atoi(argv[4]));
    else
        srand((unsigned)time(NULL));

    // Allocate space for the matricies
    A = (float*) malloc(N * N * sizeof(float));
    B = (float*) malloc(N * N * sizeof(float));

    // Initialize the matrix with random values
    for (int row = 0; row < N; row++)
        for (int col = 0; col < N; col++) {
            A[row * N + col] = (float)rand() / 32768.0;
            B[row * N + col] = 0.0;
        }
}

/// Print a matrix's content for debugging
void print_matrix(float* m) {
    printf("[");
    for (int row = 0; row < N; row++) {
        printf("\n\t");
        for (int col = 0; col < N; col++)
            printf("%10.5f\t", m[row * N + col]);
    }
    printf("]\n");
}

 /* Kernel function */
__global__ void matrixNorm(float* A, float* B, int N) {
    int row;         // Row index for loops
    float mu, sigma; // Mean and Standard Deviation

    // Calculate column number
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Maybe some wasted threads
    if (col < N) {
        // Calculate mean for column
        mu = 0.0;
        for (row = 0; row < N; ++row)
            mu += A[row * N + col];
            mu /= N;
        __syncthreads();

        // Calculate standard deviation for the column
        sigma = 0.0;
        for (row = 0; row < N; ++row)
            sigma += powf(A[col * N + row] - mu, 2.0);
        sigma /= N;
        sigma = sqrt(sigma);
        __syncthreads();

        // Normalize column
        for (row = 0; row < N; ++row)
            if (sigma == 0.0)
                B[row * N + col] = 0.0;
            else
                B[row * N + col] = (A[col * N + row] - mu) / sigma;
    }
    __syncthreads();
}

int main(int argc, char** argv) {
    /* Timing variables */
    struct timeval start, stop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    unsigned long long runtime;

    /* Initialize A and B */
    float* A, * B;
    initialize_inputs(argc, argv, A, B);

    /* Start Clock */
    gettimeofday(&start, &tzdummy);

    // Create buffers on device
    float* gpu_A, * gpu_B;
    hipMalloc((void**) &gpu_A, N * N * sizeof(float));
    hipMalloc((void**) &gpu_B, N * N * sizeof(float));

    // Send problem to device
    hipMemcpy((void*) gpu_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*) gpu_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    /* Matrix Normalization */
    matrixNorm<<<n_blocks, n_threads_per_block>>>(gpu_A, gpu_B, N);

    // This should get overwritten
    B[1] = 1233456.12345;

    // Pull result from the device
    hipMemcpy((void*) B, gpu_B, N * N * sizeof(float), hipMemcpyDeviceToHost);

    /* Stop Clock */
    gettimeofday(&stop, &tzdummy);
    runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);

    /* Display timing results */
    printf("Runtime = %g ms.\n", (float)runtime/(float)1000);

    // Debug for small N
    if (N <= 20) {
        print_matrix(A);
        print_matrix(B);
    }

    // Cleanup and exit
    // free(A);
    // free(B);
    return 0;
}
